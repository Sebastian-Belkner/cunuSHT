
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <cmath>


__device__ double dev_power_element(double value, int exponent){
    double result = exponent > 1 ? value : 1;
    for (int i = 1; i < exponent; i++) {
        result *= value;
    }
    return result;
}

__device__ double* dev_sin(double* arr, const int size){
    double* result = (double*)malloc(size * sizeof(double));
    for (int i = 0; i < size; i++) {
        result[i] = sin(arr[i]);
    }
    return result;
}

__device__ void maximum(const double* arr1, const double* arr2, double* result, const int size) {
    for (int i = 0; i < size; i++) {
        result[i] = arr1[i] > arr2[i] ? arr1[i] : arr2[i];
    }
}

__device__ double* dev_cos(double* arr, const int size){
    double* result = (double*)malloc(size * sizeof(double));
    for (int i = 0; i < size; i++) {
        result[i] = cos(arr[i]);
    }
    return result;
}

__device__ double* dev_asin(double* arr, const int size){
    double* result = (double*)malloc(size * sizeof(double));
    for (int i = 0; i < size; i++) {
        result[i] = asin(arr[i]);
    }
    return result;
}

__device__ double* dev_acos(double* arr, const int size){
    double* result = (double*)malloc(size * sizeof(double));
    for (int i = 0; i < size; i++) {
        result[i] = acos(arr[i]);
    }
    return result;
}

__device__ double* dev_atan2(double* arr, double* arr2, const int size){
    double* result = (double*)malloc(size * sizeof(double));
    for (int i = 0; i < size; i++) {
        result[i] = atan2(arr[1], arr2[i]);
    }
    return result;
}

__device__ int* ring2pixs(int ringstart, int nphi) {
    int* concatenated = (int*)malloc(nphi * sizeof(int));
    for (int i = 0; i < nphi; i++) {
        concatenated[i] = ringstart + i;
    }
    return concatenated;
}

__device__ double* dev_besselj0(double* x, const int size) {
    double* result = (double*)malloc(size * sizeof(double));
    double sum, factorial, power, term;
    for (int i = 0; i < size; i++) { 
        sum = 1.0;
        factorial = 1.0;
        power = 1.0;
        for (int k = 1; k < 50; k++) {
            factorial *= k;
            power *= (x[i] / 2.0) * (x[i] / 2.0);
            term = power / (factorial * factorial);
            sum += term * term;
        }
        result[i] = sum;
    }
    return result;
}

__device__ double* getelements(double* arr, int* indices, const int size_indices){
    double* result = (double*)malloc(size_indices * sizeof(double));
    for (int i = 0; i < size_indices; i++) {
        result[i] = arr[indices[i]];
    }
    return result;
}

__device__ double* sindod_m1(double* d, const int size){
    double* result = (double*)malloc(size * sizeof(double));
    double* d2 = (double*)malloc(size * sizeof(double));
    for (int i = 0; i < size; i++) {
        d2[i] = d[i] * d[i];
    }
    for (int i = 0; i < size; i++) {
        result[i] = -1./6. * dev_power_element(d2[i],2) + 1./120. * dev_power_element(d2[i],4) - 1./5040. * dev_power_element(d2[i],6);
    }
    return result;
}

__device__ double* dev_norm(double* x, double* y, const int size) {
    double* result = (double*)malloc(size * sizeof(double));
    for (int i = 0; i < size; i++) {
        result[i] = sqrt(x[i] * x[i] + y[i] * y[i]);
    }
    return result;
}

__device__ double dev_max(const double* arr, int size) {
    double max = arr[0];
    for (int i = 1; i < size; i++) {
        if (arr[i] > max) {
            max = arr[i];
        }
    }
    return max;
}

__device__ double* d2ang(double* red, double* imd, double* tht, double* phi, int triquand){
    const int npix = sizeof(red) / sizeof(red[0]);
    double *d = dev_norm(red, imd, npix);
    double* sind_d = (double*)malloc(npix * sizeof(double));
    double PI = 3.14159265359;
    double *dphi = (double*)malloc(npix * sizeof(double));
    double *thtp = (double*)malloc(npix * sizeof(double));
    if (dev_max(d, npix) > 0.01){
        sind_d = dev_besselj0(d, npix);
    } else {
        double *buffer = sindod_m1(d, npix);
        for (int i = 0; i < npix; i++) {
            sind_d[i] = 1. + buffer[i]; // # sin(d) / d avoiding division by zero or near zero, assuming small deflections
        }
    }
    if (triquand == 0){ // #---'close' to equator, where cost ~ 0
        double *cost = dev_cos(tht, npix);
        double *cosd = dev_cos(d, npix);
        for (int i = 0; i < npix; i++) {
            if (cost[i] > 0.8) {
                printf("wrong localization: %f\n", cost[i]);
            }
        }
        double *costp = (double*)malloc(npix * sizeof(double));

        
        double *bufferasin = (double*)malloc(npix * sizeof(double));
        for (int i = 0; i < npix; i++) {
            bufferasin[i] = imd[i] / sqrt(1. - dev_power_element(cost[i],2)) * sind_d[i];
        }
        dphi = dev_asin(bufferasin, npix);
        for (int i = 0; i < npix; i++) {
            costp[i] = cost[i] * cosd[i] - red[i] * sind_d[i] * sqrt(1. - dev_power_element(cost[i],2));
        }
        thtp = dev_acos(costp, npix);
    } else {
        int isnorth = triquand == 1 ? 1 : 0;
        double *sint = dev_sin(tht, npix);
        double ththalf[npix] = {0.0};
        for (int i = 0; i < npix; i++) {
            ththalf[i] = tht[i] * 0.5;
        }
        double *e_t = isnorth == 1 ? dev_sin(ththalf, npix) : dev_cos(ththalf, npix);  //# 1 -+ costh with no precision loss
        for (int i = 0; i < npix; i++) {
            e_t[i] = 2. * dev_power_element(e_t[i],2);
        }
        double dhalf[npix] = {0.0};
        for (int i = 0; i < npix; i++) {
            dhalf[i] = d[i] * 0.5;
        }
        double* e_d = dev_sin(dhalf, npix);
        for (int i = 0; i < npix; i++) {
            e_d[i] = 2. * dev_power_element(e_d[i],2);
        }
        double e_tp[npix] = {0.0};
        for (int i = 0; i < npix; i++) {
            e_tp[i] = e_t[i] + e_d[i] - e_t[i] * e_d[i] + triquand * red[i] * sind_d[i] * sint[i];  //# 1 -+ cost'
        }
        double max[npix] = {0.0};
        double zeros[npix] = {0.0};
        double factor[npix] = {0.0};
        for (int i = 0; i < npix; i++) {
            factor[i] = e_tp[i] * (2. - e_tp[i]);
        }
        maximum(zeros, factor, max, npix);
        double sintp[npix];
        for (int i = 0; i < npix; i++) {
            sintp[i] = sqrt(max[i]);
        }
        if (isnorth == 1){
            //assert np.max(tht) < np.pi * 0.4, ('wrong localization', np.max(tht)); //# -- for the arcsin at the end
            thtp = dev_asin(sintp, npix);
            double buffer_x[npix], buffer_y[npix]; 
            for (int i = 0; i < npix; i++) {
                // TODO possible x/y confusion
                buffer_x[i]  = (1. - e_d[i]) * sint[i] + red[i] * sind_d[i] * (1. - e_t[i]);
                buffer_y[i] = imd[i] * sind_d[i];
            dphi = dev_atan2(buffer_y, buffer_x, npix);
            }
        } else {
            //assert np.min(tht) > np.pi * 0.4, ('wrong localization', np.min(tht)); //# -- for the arcsin at the end
            thtp = dev_asin(sintp, npix);
            double buffer_x[npix], buffer_y[npix]; 
            for (int i = 0; i < npix; i++) {
                // TODO possible x/y confusion
                thtp[i] = PI -  thtp[i];
                buffer_x[i]  = (1. - e_d[i]) * sint[i] + red[i] * sind_d[i] * (e_t[i] - 1.);
                buffer_y[i] = imd[i] * sind_d[i];
            }
            dphi = dev_atan2(buffer_y, buffer_x, npix);
        }
    }
    double *ret = (double*)malloc(2*npix * sizeof(double));
    for (int i = 0; i < npix; i++) {
        ret[i] = thtp[i];
        ret[i + npix] = fmod(phi[i] + dphi[i], 2. * PI);
    }
    return ret;
}

__device__ int dev_gettriquand(double theta){
    //"""Returns the version of the pointing computation"""
    return round(cos(theta)+0.5);
}

__device__ int* dev_arange(int start, int end){
    int size = (end - start);
    int* res = (int*)malloc(size * sizeof(int));
    for (int i = 0; i < size; i++) {
        res[i] = start + i;
    }
    return res;
}

__global__ void compute_pointing(double* thetas, double* phi0, int* nphis, double* ringstarts, double* red, double* imd, int nrings, double *pointings) {
    //idx is nrings
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double PI = 3.14159265359;
    if (idx <= nrings) {
        int ringstart = ringstarts[idx];
        const int npixs = nphis[idx];
        int* pixs = ring2pixs(ringstart, npixs);
        
        if (npixs > 0) {
            double* t_red = getelements(red, pixs, npixs);
            double* i_imd = getelements(imd, pixs, npixs);

            double *phis = (double*)malloc(npixs * sizeof(double));
            for (int i = 0; i < npixs; i++) {
                phis[i] = fmod(phi0[idx] + i * (2. * PI / npixs),2. * PI);
            }
            //TODO implement this for correct offset
            // phis = phis//[pixs - self.geom.ofs[ir]]
            double *thts = (double*)malloc(npixs * sizeof(double));
            for (int i = 0; i < npixs; i++) {
                thts[i] = thetas[idx];
            }
            double *buff = d2ang(t_red, i_imd, thts, phis, dev_gettriquand(thetas[idx]));
            int *sli = dev_arange(ringstart, ringstart + npixs);
            for (int i = 0; i < npixs; i++) {
                int idx_  = sli[i];
                pointings[idx + idx_] = buff[i];
                pointings[idx + 1 + idx_ + npixs] = buff[i + npixs];
                // TODO implement this (rotation of the polarization angle)
                // cot = np.cos(self.geom.theta[ir]) / np.sin(self.geom.theta[ir])
                // d = np.sqrt(t_red ** 2 + i_imd ** 2)
                // thp_phip_gamma[2, sli] = np.arctan2(i_imd, t_red ) - np.arctan2(i_imd, d * np.sin(d) * cot + t_red * np.cos(d))
                // startpix += len(pixs)
            }
        }
    }
}

bool allGreaterThanZero(double* array, int size) {
    for (int i = 0; i < size; i++) {
        if (array[i] <= 0.0) {
            return false;
        }
    }
    return true;
}

bool allLessThanPi(double* array, int size) {
    double PI = 3.14159265359;
    for (int i = 0; i < size; i++) {
        if (array[i] >= PI) {
            return false;
        }
    }
    return true;
}

int sum(int* array, int size) {
    printf("size: %d\n", size);
    int result = 0;
    for (int i = 0; i < size; i++) {
        result += array[i];
    }
    return result;
}

int* argsort(double* array, int size) {
    int* indices = (int*)malloc(size * sizeof(int));
    if (indices == NULL) {
        fprintf(stderr, "Memory allocation failed\n");
        exit(EXIT_FAILURE);
    }

    for (int i = 0; i < size; i++) {
        indices[i] = i;
    }
    for (int i = 0; i < size - 1; i++) {
        for (int j = 0; j < size - i - 1; j++) {
            if (array[indices[j]] > array[indices[j + 1]]) {
                // Swap indices
                int temp = indices[j];
                indices[j] = indices[j + 1];
                indices[j + 1] = temp;
            }
        }
    }
    return indices;
}

extern "C" void pointing(double* thetas, double* phi0, int* nphis, double *ringstarts, double *red, double *imd, int nrings, int npix, double *host_result) {
    double *device_thetas, *device_phi0,  *device_ringstarts, *device_red, *device_imd;
    int *device_nphis;
    double *device_result;

    int startpix = 0;
    bool condition1 = allGreaterThanZero(thetas, npix);
    bool condition2 = allLessThanPi(thetas, npix);
    printf("nphis:");
    for (int i = 0; i < 2048; i++) {
        printf("%d ", nphis[i]);
    }
    printf("condition1: %d\n", condition1);
    printf("condition2: %d\n", condition2);
    printf("npix: %d\n", npix);
    // assert(condition1 && condition2);

    int* sorted_ringstarts = argsort(ringstarts, nrings);

    hipMalloc((void**)&device_thetas, nrings * sizeof(double));
    hipMalloc((void**)&device_phi0, nrings * sizeof(double));
    hipMalloc((void**)&device_nphis, nrings * sizeof(int));
    hipMalloc((void**)&device_ringstarts, nrings * sizeof(double));
    hipMalloc((void**)&device_red, npix * sizeof(double));
    hipMalloc((void**)&device_imd, npix * sizeof(double));

    hipMalloc((void**)&device_result, 2*npix * sizeof(double));

    hipMemcpy(device_thetas, thetas, nrings * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_phi0, phi0, nrings * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_nphis, nphis, nrings * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_ringstarts, ringstarts, nrings * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_red, red, npix * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_imd, imd, npix * sizeof(double), hipMemcpyHostToDevice);

    // auto start = high_resolution_clock::now();
    // const int threadsPerBlock = 256;
    // const int blocksPerGrid = (lmax + threadsPerBlock - 1) / threadsPerBlock;
    compute_pointing<<<1, nrings>>>(device_thetas, device_phi0, device_nphis, device_ringstarts, device_red, device_imd, nrings, device_result);
    hipDeviceSynchronize();
    // auto stop = high_resolution_clock::now();

    hipMemcpy(host_result, device_result, 2*npix * sizeof(double), hipMemcpyDeviceToHost);

    // for (int l = lmax; l <= lmax; ++l) {
        // for (int m = 0; m <= l; ++m) {

    // printf("P_%d^%d(x): ",lmax,mmax);
    // // printf("%d", nrings);
    // for (int i = 0; i < nrings; ++i) {
    //     printf("%.2f ", host_result[i]);
    // }
    // printf("\n");
        // }
    // }

    hipFree(device_thetas);
    hipFree(device_phi0);
    hipFree(device_nphis);
    hipFree(device_ringstarts);
    hipFree(device_red);
    hipFree(device_imd);
    hipFree(device_result);
}