#include "hip/hip_runtime.h"
#include <nanobind/nanobind.h>
#include <nanobind/ndarray.h>
#include <nanobind/stl/array.h>
#include <nanobind/stl/tuple.h>

#include <stdio.h>
#include <stdlib.h>

#include <math.h>
#include <assert.h>
#include <cmath>
#include <chrono>
#include <time.h>

// #include "pointing.h"
#include "kernel_params.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>

namespace nb = nanobind;
using namespace nb::literals;


// void float_to_double(const float* src, double* dest, int size) {
//     for (int i = 0; i < size; i++) {
//         dest[i] = (double)src[i];
//     }
// }

// std::tuple<intptr_t, intptr_t> Cpointing_ptrs(intptr_t thetas_, intptr_t phi0_, intptr_t nphis_, intptr_t ringstarts_, intptr_t synthmap_, int nring, int npix, double *host_result){
//     printf("pointing.cpp:: Cpointing_ptrs\n");
//     KernelParams kp;
//     kp.thetas = reinterpret_cast<double*>(thetas_);
//     kp.phi0 = reinterpret_cast<double*>(phi0_);
//     kp.nphis = reinterpret_cast<int*>(nphis_);
//     kp.ringstarts = reinterpret_cast<int*>(ringstarts_);
//     kp.synthmap = reinterpret_cast<double*>(synthmap_);
//     kp.nring = nring;
//     kp.npix = npix;

//     auto devres = CUpointing_struct(kp);
//     return std::make_tuple(reinterpret_cast<intptr_t>(std::get<0>(devres)),reinterpret_cast<intptr_t>(std::get<1>(devres)));
// }


// void CUfloat_to_double(const float* src, double* dest, int size) {
//     for (int i = 0; i < size; i++) {
//         dest[i] = (double)src[i];
//     }
// }

__device__ double dev_power_element(double value, int exponent){
    double result = exponent > 1 ? value : 1;
    for (int i = 1; i < exponent; i++) {
        result *= value;
    }
    return result;
}

__device__ void dev_besselj0(double* x, const int start, const int size, double* result) {
    double factorial, power, term;
    for (int i = start; i < size; i++) { 
        factorial = 1.0;
        power = 1.0;
        for (int k = 1; k < 25; k++) {
            factorial *= k;
            power *= (x[i] / 2.0) * (x[i] / 2.0);
            term = power / (factorial * factorial);
            result[i] += term * term;
        }
    }
}

__device__ void sindod_m1(double* d, int start, int size, double* result){
    for (int i = start; i < size; i++) {
        result[i] = 1. + -1./6. * d[i] * (1. - 1./20. * d[i] *(1. - 1./42. * d[i]));
        // result[i] = 1. + (-1./6. * d[i]*d[i] + 1./120. * d[i]*d[i]*d[i]*d[i] - 1./5040. * d[i]*d[i]*d[i]*d[i]*d[i]*d[i]);
    }
}

__device__ void dev_norm2(double* x, double* y, const int start, const int size, double* result) {
    for (int i = start; i < size; i++) {
        result[i] = x[i] * x[i] + y[i] * y[i];
    }
}

__device__ void dev_norm(double* x, double* y, const int start, const int size, double* result) {
    for (int i = start; i < size; i++) {
        result[i] = sqrt(x[i] * x[i] + y[i] * y[i]);
    }
}

__device__ int dev_isbigger(const double* arr, const int start, int size, const double threshold) {
    for (int i = start; i < size; i++) {
        if (arr[i] > threshold) {
            return 1;
        }
    }     return 0;
}

__device__ int dev_gettriquand(double theta){
    //"""Returns the version of the pointing computation"""
    return round(cos(theta)+0.5);
}

__device__ int* dev_arange(int start, int end){
    int size = (end - start);
    int* res = (int*)malloc(size * sizeof(int));
    for (int i = 0; i < size; i++) {
        res[i] = start + i;
    }
    return res;
}

template <typename Scalar>
__global__ void compute_dummy(Scalar* pt, Scalar* pp, const Scalar* thetas, const Scalar* phi0, const size_t* nphis, const size_t* ringstarts, const Scalar* synthmap, const size_t nring, const size_t npix, KernelLocals kl, const size_t size) {
    //idx is nring
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double PI = 3.14159265359;
    if (1 == 1) {
        if (idx <= nring) {
            pt[idx] = synthmap[idx];
            pp[idx] = synthmap[idx];
        }
    }
}

template <typename Scalar>
__global__ void compute_pointing_1Dto1D(Scalar* pt, Scalar* pp, const Scalar* thetas, const Scalar* phi0, const size_t* nphis, const size_t* ringstarts, const Scalar* spin1_theta, const Scalar* spin1_phi, const size_t nring, const size_t npix, KernelLocals kl, const size_t size) {
    //idx is nring
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double PI = 3.14159265359;
    if (idx <= nring) {
        const int ringstart = ringstarts[idx];
        const int npixring = nphis[idx];
        // kl.e_r(sin(theta(iring)), 0, cos(theta(iring)));
        kl.sint = sin(thetas[idx]); 
        kl.cost = cos(thetas[idx]);
        for (int i = ringstart; i < ringstart+npixring; i++) {
            kl.phi[i] = phi0[idx] + i * (2. * PI / npixring);
        }
        for (int i = ringstart; i < ringstart+npixring; i++) {
            kl.d[i] = spin1_theta[i] * spin1_theta[i] + spin1_phi[i] * spin1_theta[i+npix];
        }
        
        if (dev_isbigger(kl.d, ringstart, ringstart+npixring, 0.001)){
            for (int i = ringstart; i < ringstart+npixring; i++) {
                kl.a[i] = sqrt(kl.d[i]);
                kl.sind_d[i] = sin(kl.a[i]) / kl.a[i];
                kl.cos_a[i] = cos(kl.a[i]);
                kl.twohav_aod[i] = (kl.cos_a[i] - 1.) / kl.d[i];
            }
            
        } else {
            sindod_m1(kl.d, ringstart, ringstart+npixring, kl.sind_d);
            for (int i = ringstart; i < ringstart+npixring; i++) {
                kl.twohav_aod[i] = -0.5 + kl.d[i]/24. * (1. - kl.d[i]/30. * (1. - kl.d[i]/56.));
                kl.cos_a[i] = 1. + kl.d[i] * kl.twohav_aod[i];
            }
        }
        for (int i = ringstart; i < ringstart+npixring; i++) {
            kl.e_a1[i] = kl.cost * spin1_theta[i];
            kl.e_a2[i] = kl.phi[i];
            kl.e_a3[i] = -kl.sint * spin1_phi[i];
        }

        // kl.n_prime(kl.e_r * kl.cos_a + kl.e_a * kl.sin_d);
        for (int i = ringstart; i < ringstart+npixring; i++) {
            kl.np1[i] = kl.sint * kl.cos_a[i] + kl.e_a1[i] * kl.sind_d[i];
            kl.np2[i] = 0. - kl.e_a2[i] * kl.sind_d[i];
            kl.np3[i] = kl.cost * kl.cos_a[i] + kl.e_a3[i] * kl.sind_d[i];
        }

        //theta = std::atan2(sqrt(inp.x*inp.x+inp.y*inp.y),inp.z);
        //phi = safe_atan2 (inp.y,inp.x);
        //if (phi<0.) phi += twopi;
        for (int i = ringstart; i < ringstart+npixring; i++) {
            kl.npt[i] = atan2(sqrt(kl.np1[i]*kl.np1[i] + kl.np2[i] * kl.np2[i]), kl.np3[i]);
            kl.npp[i] = atan2(kl.np2[i], kl.np1[i]);
            kl.npp[i] = (kl.npp[i] < 0.) ? (kl.npp[i] + 2.*PI) : kl.npp[i];
        }
        
        // kl.phinew = (kl.phinew >= 2.*PI) ? (kl.phinew - 2.*PI) : kl.phinew;
        for (int i = ringstart; i < ringstart+npixring; i++) {
            pt[i] = kl.npt[i];
            pp[i] = kl.npp[i] + kl.phi[i];
            pp[i] = (pp[i] >= 2*PI) ? (pp[i] - 2.*PI) : pp[i];
        }
    }
}

template <typename Scalar>
__global__ void compute_pointing_cparr(Scalar* pt, Scalar* pp, const Scalar* thetas, const Scalar* phi0, const size_t* nphis, const size_t* ringstarts, const Scalar* synthmap, const size_t nring, const size_t npix, KernelLocals kl, const size_t size) {
    //idx is nring
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double PI = 3.14159265359;
    if (idx <= nring) {
        const int ringstart = ringstarts[idx];
        const int npixring = nphis[idx];
        // kl.e_r(sin(theta(iring)), 0, cos(theta(iring)));
        kl.sint = sin(thetas[idx]); 
        kl.cost = cos(thetas[idx]);
        for (int i = ringstart; i < ringstart+npixring; i++) {
            kl.phi[i] = phi0[idx] + i * (2. * PI / npixring);
        }
        for (int i = ringstart; i < ringstart+npixring; i++) {
            kl.d[i] = synthmap[i] * synthmap[i] + synthmap[i+npix] * synthmap[i+npix];
        }
        
        if (dev_isbigger(kl.d, ringstart, ringstart+npixring, 0.001)){
            for (int i = ringstart; i < ringstart+npixring; i++) {
                kl.a[i] = sqrt(kl.d[i]);
                kl.sind_d[i] = sin(kl.a[i]) / kl.a[i];
                kl.cos_a[i] = cos(kl.a[i]);
                kl.twohav_aod[i] = (kl.cos_a[i] - 1.) / kl.d[i];
            }
            
        } else {
            sindod_m1(kl.d, ringstart, ringstart+npixring, kl.sind_d);
            for (int i = ringstart; i < ringstart+npixring; i++) {
                kl.twohav_aod[i] = -0.5 + kl.d[i]/24. * (1. - kl.d[i]/30. * (1. - kl.d[i]/56.));
                kl.cos_a[i] = 1. + kl.d[i] * kl.twohav_aod[i];
            }
        }
        for (int i = ringstart; i < ringstart+npixring; i++) {
            kl.e_a1[i] = kl.cost * synthmap[i];
            kl.e_a2[i] = kl.phi[i];
            kl.e_a3[i] = -kl.sint * synthmap[i];
        }

        // kl.n_prime(kl.e_r * kl.cos_a + kl.e_a * kl.sin_d);
        for (int i = ringstart; i < ringstart+npixring; i++) {
            kl.np1[i] = kl.sint * kl.cos_a[i] + kl.e_a1[i] * kl.sind_d[i];
            kl.np2[i] = 0. - kl.e_a2[i] * kl.sind_d[i];
            kl.np3[i] = kl.cost * kl.cos_a[i] + kl.e_a3[i] * kl.sind_d[i];
        }

        //theta = std::atan2(sqrt(inp.x*inp.x+inp.y*inp.y),inp.z);
        //phi = safe_atan2 (inp.y,inp.x);
        //if (phi<0.) phi += twopi;
        for (int i = ringstart; i < ringstart+npixring; i++) {
            kl.npt[i] = atan2(sqrt(kl.np1[i]*kl.np1[i] + kl.np2[i] * kl.np2[i]), kl.np3[i]);
            kl.npp[i] = atan2(kl.np2[i], kl.np1[i]);
            kl.npp[i] = (kl.npp[i] < 0.) ? (kl.npp[i] + 2.*PI) : kl.npp[i];
        }
        
        // kl.phinew = (kl.phinew >= 2.*PI) ? (kl.phinew - 2.*PI) : kl.phinew;
        for (int i = ringstart; i < ringstart+npixring; i++) {
            pt[i] = kl.npt[i];
            pp[i] = kl.npp[i] + kl.phi[i];
            pp[i] = (pp[i] >= 2*PI) ? (pp[i] - 2.*PI) : pp[i];
        }
    }
}

template <typename Scalar>
void CUpointing_1Dto1D(
    nb::ndarray<const Scalar, nb::ndim<1>, nb::device::cuda> thetas,
    nb::ndarray<const Scalar, nb::ndim<1>, nb::device::cuda> phi0,
    nb::ndarray<const size_t, nb::ndim<1>, nb::device::cuda> nphis,
    nb::ndarray<const size_t, nb::ndim<1>, nb::device::cuda> ringstarts,
    nb::ndarray<const Scalar, nb::ndim<1>, nb::device::cuda> spin1_theta,
    nb::ndarray<const Scalar, nb::ndim<1>, nb::device::cuda> spin1_phi,
    nb::ndarray<Scalar, nb::ndim<1>, nb::device::cuda> outarr_pt,
    nb::ndarray<Scalar, nb::ndim<1>, nb::device::cuda> outarr_pp) {

    const size_t size = thetas.size();
    const size_t npix = spin1_theta.size();
    const size_t nring = ringstarts.size();
    size_t block_size = 256;
    size_t num_blocks = (size + block_size - 1) / block_size;

    KernelLocals kl;
    double *dev_sint;
    double *dev_cost;
    double *dev_phi;
    double *dev_sind_d, *dev_a, *dev_d;
    double *dev_cos_a, *dev_twohav_aod;
    double *dev_e_a1, *dev_e_a2, *dev_e_a3;
    double *dev_np1, *dev_np2, *dev_np3;
    double *dev_npt, *dev_npp;
    double* dev_philocs;

    hipMalloc((void**)&dev_phi, npix * sizeof(double));
    hipMalloc((void**)&dev_sind_d, npix * sizeof(double));
    hipMalloc((void**)&dev_a, npix * sizeof(double));
    hipMalloc((void**)&dev_d, npix * sizeof(double));
    hipMalloc((void**)&dev_cos_a, npix * sizeof(double));
    hipMalloc((void**)&dev_twohav_aod, npix * sizeof(double));
    hipMalloc((void**)&dev_e_a1, npix * sizeof(double));
    hipMalloc((void**)&dev_e_a2, npix * sizeof(double));
    hipMalloc((void**)&dev_e_a3, npix * sizeof(double));
    hipMalloc((void**)&dev_np1, npix * sizeof(double));
    hipMalloc((void**)&dev_np2, npix * sizeof(double));
    hipMalloc((void**)&dev_np3, npix * sizeof(double));
    hipMalloc((void**)&dev_npt, npix * sizeof(double));
    hipMalloc((void**)&dev_npp, npix * sizeof(double));
    hipMalloc((void**)&dev_philocs, npix * sizeof(double));


    kl.phi = dev_phi;
    kl.sind_d = dev_sind_d;
    kl.a = dev_a;
    kl.d = dev_d;
    kl.cos_a = dev_cos_a;
    kl.twohav_aod = dev_twohav_aod;
    kl.e_a1 = dev_e_a1;
    kl.e_a2 = dev_e_a2;
    kl.e_a3 = dev_e_a3;
    kl.np1 = dev_np1;
    kl.np2 = dev_np2;
    kl.np3 = dev_np3;
    kl.npt = dev_npt;
    kl.npp = dev_npp;
    kl.philocs = dev_philocs;
    hipDeviceSynchronize();
    // compute_dummy<<<num_blocks, block_size>>>(outarr_pt.data(), outarr_pp.data(), thetas.data(), phi0.data(), nphis.data(), ringstarts.data(), synthmap.data(), nring, npix, kl, size);
    compute_pointing_1Dto1D<<<num_blocks, block_size>>>(outarr_pt.data(), outarr_pp.data(), thetas.data(), phi0.data(), nphis.data(), ringstarts.data(), spin1_theta.data(), spin1_phi.data(), nring, npix, kl, size);
    hipDeviceSynchronize();
}

template <typename Scalar>
void CUpointing_cparr(
    nb::ndarray<const Scalar, nb::ndim<1>, nb::device::cuda> thetas,
    nb::ndarray<const Scalar, nb::ndim<1>, nb::device::cuda> phi0,
    nb::ndarray<const size_t, nb::ndim<1>, nb::device::cuda> nphis,
    nb::ndarray<const size_t, nb::ndim<1>, nb::device::cuda> ringstarts,
    nb::ndarray<const Scalar, nb::ndim<1>, nb::device::cuda> synthmap,
    nb::ndarray<Scalar, nb::ndim<1>, nb::device::cuda> outarr_pt,
    nb::ndarray<Scalar, nb::ndim<1>, nb::device::cuda> outarr_pp) {

    const size_t size = thetas.size();
    const size_t npix = synthmap.size()/2;
    const size_t nring = ringstarts.size();
    size_t block_size = 256;
    size_t num_blocks = (size + block_size - 1) / block_size;

    KernelLocals kl;
    double *dev_sint;
    double *dev_cost;
    double *dev_phi;
    double *dev_sind_d, *dev_a, *dev_d;
    double *dev_cos_a, *dev_twohav_aod;
    double *dev_e_a1, *dev_e_a2, *dev_e_a3;
    double *dev_np1, *dev_np2, *dev_np3;
    double *dev_npt, *dev_npp;
    double* dev_philocs;

    hipMalloc((void**)&dev_phi, npix * sizeof(double));
    hipMalloc((void**)&dev_sind_d, npix * sizeof(double));
    hipMalloc((void**)&dev_a, npix * sizeof(double));
    hipMalloc((void**)&dev_d, npix * sizeof(double));
    hipMalloc((void**)&dev_cos_a, npix * sizeof(double));
    hipMalloc((void**)&dev_twohav_aod, npix * sizeof(double));
    hipMalloc((void**)&dev_e_a1, npix * sizeof(double));
    hipMalloc((void**)&dev_e_a2, npix * sizeof(double));
    hipMalloc((void**)&dev_e_a3, npix * sizeof(double));
    hipMalloc((void**)&dev_np1, npix * sizeof(double));
    hipMalloc((void**)&dev_np2, npix * sizeof(double));
    hipMalloc((void**)&dev_np3, npix * sizeof(double));
    hipMalloc((void**)&dev_npt, npix * sizeof(double));
    hipMalloc((void**)&dev_npp, npix * sizeof(double));
    hipMalloc((void**)&dev_philocs, npix * sizeof(double));


    kl.phi = dev_phi;
    kl.sind_d = dev_sind_d;
    kl.a = dev_a;
    kl.d = dev_d;
    kl.cos_a = dev_cos_a;
    kl.twohav_aod = dev_twohav_aod;
    kl.e_a1 = dev_e_a1;
    kl.e_a2 = dev_e_a2;
    kl.e_a3 = dev_e_a3;
    kl.np1 = dev_np1;
    kl.np2 = dev_np2;
    kl.np3 = dev_np3;
    kl.npt = dev_npt;
    kl.npp = dev_npp;
    kl.philocs = dev_philocs;
    // compute_dummy<<<num_blocks, block_size>>>(outarr_pt.data(), outarr_pp.data(), thetas.data(), phi0.data(), nphis.data(), ringstarts.data(), synthmap.data(), nring, npix, kl, size);
    compute_pointing_cparr<<<num_blocks, block_size>>>(outarr_pt.data(), outarr_pp.data(), thetas.data(), phi0.data(), nphis.data(), ringstarts.data(), synthmap.data(), nring, npix, kl, size);
    hipDeviceSynchronize();

    hipError_t errSync  = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess) 
    printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
    printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
}

NB_MODULE(popy, m) {
    // m.def(
    //     "Cpointing_ptrs",
    //     [](
    //         intptr_t thetas_,
    //         intptr_t phi0_,
    //         intptr_t nphis_,
    //         intptr_t ringstarts_,
    //         intptr_t synthmap_,
    //         int nring,
    //         int npix,
    //         nb::ndarray<double>&host_result
    //         ) {
    //         return Cpointing_ptrs(thetas_, phi0_, nphis_, ringstarts_, synthmap_, nring, npix, host_result.data());
    //     }
    // );
    m.def("CUpointing_1Dto1D",
        &CUpointing_1Dto1D<double>,
        "thetas"_a.noconvert(),
        "phi0"_a.noconvert(),
        "nphis"_a.noconvert(),
        "ringstarts"_a.noconvert(),
        "spin1_theta"_a.noconvert(),
        "spin1_phi"_a.noconvert(),
        "outarr_pt"_a.noconvert(),
        "outarr_pp"_a.noconvert()
    );
    m.def("CUpointing_cparr",
        &CUpointing_cparr<double>,
        "thetas"_a.noconvert(),
        "phi0"_a.noconvert(),
        "nphis"_a.noconvert(),
        "ringstarts"_a.noconvert(),
        "synthmap"_a.noconvert(),
        "outarr_pt"_a.noconvert(),
        "outarr_pp"_a.noconvert()
    );
}