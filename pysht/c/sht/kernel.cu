
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
// #include <cuda_runtime.h>


__device__ int fibonaccidown(int n) {
    if (n <= 1) {
        return n;
    }
    return fibonaccidown(n - 1) + fibonaccidown(n - 2);
}


__global__ void fibonaccidown_kernel(int n, int *result) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        result[tid] = fibonaccidown(tid);
    }
}

__device__ int fibonacciup(int nmax, int counter, int fib0, int fib1) {
    if (counter==nmax) {
        if (nmax==0) {
            return 1;
        } else if (nmax==1) {
            return 1;
        }
        return fib1;
     } else if (counter < nmax) {
            return fibonacciup(nmax, counter+1, fib1, fib0+fib1);  
     }
}


__global__ void fibonacciup_kernel(int n, int *result) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int counter = 1;
    if (tid < n) {
        result[tid] = fibonacciup(tid, counter, 1, 1);
    }
}


// # Wrapper function to call the CUDA kernel
extern "C" void Fibonacci(int n, int *host_result) {
    const int threadsPerBlock = 1;
    const int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    int *device_result;

    // Allocate memory on host and device
    hipMalloc((void **)&device_result, n * sizeof(int));

    // Launch kernel
    printf(" %d %d %d\n", n, threadsPerBlock, blocksPerGrid);
    // fibonaccidown_kernel<<<blocksPerGrid, threadsPerBlock>>>(n, device_result);
    fibonacciup_kernel<<<1, n>>>(n, device_result);
    hipDeviceSynchronize();

    // Copy result from device to host
    hipMemcpy(host_result, device_result, n * sizeof(int), hipMemcpyDeviceToHost);

    hipError_t errSync  = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess) 
    printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
    printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

    hipError_t cuda_error = hipGetLastError();
    if (cuda_error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(cuda_error));
    }


    printf("Fibonacci Sequence:\n");
    for (int i = 0; i < n; ++i) {
        printf("%d ", host_result[i]);
    }
    printf("\n");

    hipFree(device_result);
}


__device__ float d_legendre(int l, float x) {
    if (l == 0) {
        return 1.0f;
    } else if (l == 1) {
        return x;
    } else {
        return ((2.0f * l - 1.0f) * x * d_legendre(l - 1, x) - (l - 1) * d_legendre(l - 2, x)) / l;
    }
}
__global__ void kernel_legendre(int n, float *x, int lmax, float *result) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    if (idx < n) {
        float x_ = x[threadIdx.x];
        result[idx] = d_legendre(blockIdx.x, x_);
    }
}


// # Wrapper function to call the CUDA kernel
extern "C" void Legendre(int lmax, float *host_x, float *host_result, int size_x) {
    float *device_x, *device_result;

    // Allocate device memory
    hipMalloc((void **)&device_x, size_x * sizeof(float));
    hipMalloc((void **)&device_result, size_x * (lmax + 1) * sizeof(float));

    // Copy data from host to device
    hipMemcpy(device_x, host_x, size_x * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    // int threadsPerBlock = 256;
    // int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    printf(" %d %d\n", lmax, size_x);
    // kernel_legendre<<<blocksPerGrid, threadsPerBlock>>>(size* (lmax + 1), device_x, lmax, device_result);
    kernel_legendre<<<lmax+1, size_x>>>(size_x * (lmax + 1), device_x, lmax, device_result);
    hipDeviceSynchronize();

    // Copy result from device to host
    hipMemcpy(host_result, device_result, size_x * (lmax + 1) * sizeof(float), hipMemcpyDeviceToHost);


    // Print results
    // for (int l = 0; l <= lmax; ++l) {
    //     printf("P_%d(x) = ", l);
    //     for (int i = 0; i < size_x; ++i) {
    //         printf("%.2f ", host_result[l*size_x+i]);
    //     }
    //     printf("\n");
    // }
    // printf("done");

    // Free device memory
    hipFree(device_x);
    hipFree(device_result);
}



__global__ void multiplyByTwo(float *array, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure we stay within the array bounds
    if (tid < size) {
        array[tid] *= 2;
    }
}

// # Wrapper function to call the CUDA kernel
extern "C" void multiply(float *hostArray, float *host_output, int arraySize) {
    const int threadsPerBlock = 1024;
    float *deviceArray;

    // Allocate device memory
    hipMalloc((void **)&deviceArray, arraySize * sizeof(float));
    // Copy data from host to device
    hipMemcpy((void **)deviceArray, hostArray, arraySize * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    int blocksPerGrid = (arraySize + threadsPerBlock - 1) / threadsPerBlock;
    printf("%d %d %d\n", arraySize, threadsPerBlock, blocksPerGrid);
    multiplyByTwo<<<blocksPerGrid, threadsPerBlock>>>(deviceArray, arraySize);
    // mult<<<1, 1>>>(hostArray, arraySize);
    hipDeviceSynchronize(); // Wait for kernel to finish

    // Copy result from device to host
    hipMemcpy(host_output, deviceArray, arraySize * sizeof(float), hipMemcpyDeviceToHost);

    hipError_t errSync  = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess) 
    printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
    printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

    // Print the updated array
    printf("input \n");
    for (int i = 0; i < arraySize; ++i) {
        printf("%.2f ", hostArray[i]);
    }
    printf("\noutput \n");
    for (int i = 0; i < arraySize; ++i) {
        printf("%.2f ", host_output[i]);
    }
    printf("\n");

    // Free device memory
    hipFree(deviceArray);
}