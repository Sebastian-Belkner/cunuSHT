#include "hip/hip_runtime.h"
#include <stddef.h>

#include <nanobind/nanobind.h>
#include <nanobind/ndarray.h>
#include <nanobind/stl/array.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>

namespace nb = nanobind;
using namespace nb::literals;

template <typename Scalar>
__global__ void compute_doubling_spin0_2D(Scalar* synth2D, const size_t nring, const size_t nphi, Scalar* doublings2D) {
    // map_dfs = np.empty((2 * ntheta - 2, nphi), dtype=np.complex128 if spin == 0 else ctype[map.dtype])
    //idx is nrings
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    const size_t nphihalf = nphi / 2;
    if (idx <= nring) {
        // map_dfs[:ntheta, :] = map[0]
        for (int i = 0; i < nring; i++) {
            for (int j = 0; j < nphi; j++) {
                doublings2D[i][j] = synth2D[i][j];
            }
        }
        // map_dfs[ntheta:, :nphihalf] = map_dfs[ntheta - 2:0:-1, nphihalf:]
        // map_dfs[ntheta:, nphihalf:] = map_dfs[ntheta - 2:0:-1, :nphihalf]
        for (int i = nring; i < 2*nring; i++) {
            for (int j = 0; j < nphihalf; j++) {
                doublings2D[i][j] = synth2D[nring-i][nphihalf + j];
                doublings2D[i][nphihalf + j] = synth2D[nring-i][j];
            }
        }
    }
}

template <typename Scalar>
void CUdoubling_cparr2D(
    nb::ndarray<const Scalar, nb::ndim<2>, nb::device::cuda> synth2D,
    const size_t nring,
    const size_t nphi,
    nb::ndarray<Scalar, nb::ndim<2>, nb::device::cuda> outarr_doubling2D) {

    const int threadsPerBlock = 256;
    int blocksPerGrid = (nring + threadsPerBlock - 1) / threadsPerBlock;
    compute_doubling_spin0_2D<<<blocksPerGrid, threadsPerBlock>>>(synth2D.data(), nring, nphi, outarr_doubling2D.data());
    hipDeviceSynchronize();
}


// template <typename Scalar>
// __global__ void compute_doubling_spin0_2Dto1D(const Scalar* synth1D, const size_t ntheta, const size_t nphi, Scalar* doubling1D) {
//     // map_dfs = np.empty((2 * ntheta - 2, nphi), dtype=np.complex128 if spin == 0 else ctype[map.dtype])
//     //idx is ntheta
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;

//     const size_t nphihalf = nphi / 2;
//     if (idx <= ntheta) {
//         // map_dfs[:ntheta, :] = map[0]
//         for (int i = 0; i < ntheta; i++) {
//             for (int j = 0; j < nphi; j++) {
//                 doubling1D[i * nphi + j] = synth1D[i * nphi + j];
//             }
//         }
//         for (int i = ntheta; i < 2 * ntheta - 2; i++) {
//             for (int j = 0; j < nphihalf; j++) {
//                 doubling1D[i * nphi + j] = synth1D[(2 * ntheta - 3 - i) * nphi + (nphi - nphihalf) + j];
//             }
//         }
//         for (int i = ntheta; i < 2 * ntheta - 2; i++) {
//             for (int j = nphihalf; j < nphi; j++) {
//                 doubling1D[i * nphi + j] = synth1D[(2 * ntheta - 3 - i) * nphi + (j - nphihalf)];
//             }
//         }
//     }
// }

// template <typename Scalar>
// void CUdoubling_2Dto1D(
//     nb::ndarray<Scalar, nb::ndim<2>, nb::device::cuda> synth1D,
//     const size_t nring,
//     const size_t nphi,
//     nb::ndarray<Scalar, nb::ndim<1>, nb::device::cuda> outarr_doubling1D) {

//     const int threadsPerBlock = 256;
//     int blocksPerGrid = (nring + threadsPerBlock - 1) / threadsPerBlock;
//     compute_doubling_spin0_1D<<<blocksPerGrid, threadsPerBlock>>>(synth1D.data(), nring, nphi, outarr_doubling1D.data());
//     hipDeviceSynchronize();
// }

template <typename Scalar>
__global__ void compute_doubling_spin0_1D(const Scalar* synth1D, const size_t ntheta, const size_t nphi, Scalar* doubling1D) {
    // map_dfs = np.empty((2 * ntheta - 2, nphi), dtype=np.complex128 if spin == 0 else ctype[map.dtype])
    //idx is ntheta
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    const size_t nphihalf = nphi / 2;
    if (idx < ntheta) {
        // map_dfs[:ntheta, :] = map[0]
        for (int j = 0; j < nphi; j++) {
                doubling1D[nphi*idx+j] = synth1D[nphi*idx+j];
                // doubling1D[2*nphi*idx + j] = synth1D[nphi-j];
                // doubling1D[2*nphi*idx + j] = 2.*synth1D[j];
        }
        for (int j = 0; j < nphi/2; j++) {
            doubling1D[2*nphi*idx+j] = synth1D[nphi*idx+j];
        }
        for (int j = nphi/2; j < nphi; j++) {
            doubling1D[2*nphi*idx+j] = synth1D[nphi*idx+j-nphi/2];
        }
    }
}


template <typename Scalar>
void CUdoubling_cparr1D(
    nb::ndarray<Scalar, nb::ndim<1>, nb::device::cuda> synth1D,
    const size_t nring,
    const size_t nphi,
    nb::ndarray<Scalar, nb::ndim<1>, nb::device::cuda> outarr_doubling1D) {

    const int threadsPerBlock = 256;
    int blocksPerGrid = (nring + threadsPerBlock - 1) / threadsPerBlock;
    compute_doubling_spin0_1D<<<blocksPerGrid, threadsPerBlock>>>(synth1D.data(), nring, nphi, outarr_doubling1D.data());
    hipDeviceSynchronize();

    hipError_t errSync  = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess) 
    printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
    printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
}

NB_MODULE(dopy, m) {
    // m.def("CUdoubling_cparr2D",
    //     &CUdoubling_cparr2D<double>,
    //     "synth2D"_a.noconvert(),
    //     "nring"_a.noconvert(),
    //     "nphi"_a.noconvert(),
    //     "outarr_doubling2D"_a.noconvert()
    // );
    // m.def("CUdoubling_2Dto1D",
    //     &CUdoubling_cparr1D<double>,
    //     "synth1D"_a.noconvert(),
    //     "nring"_a.noconvert(),
    //     "nphi"_a.noconvert(),
    //     "outarr_doubling1D"_a.noconvert()
    // );
    m.def("CUdoubling_cparr1D",
        &CUdoubling_cparr1D<double>,
        "synth1D"_a.noconvert(),
        "nring"_a.noconvert(),
        "nphi"_a.noconvert(),
        "outarr_doubling1D"_a.noconvert()
    );
}